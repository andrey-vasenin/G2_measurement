#include "hip/hip_runtime.h"
﻿//
// Created by andrei on 4/13/21.
//
#include <chrono>
#include <memory>
#include <iostream>
#include <functional>
#include <vector>
#include <numeric>
#include <complex>
#include <cstdint>
#include "dsp.cuh"
#include "dsp_functors.cuh"
#include "digitizer.h"
#include "measurement.cuh"
#include "tiled_range.cuh"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include "yokogawa_gs210.h"
#include <future>
#include <thread>

namespace py = pybind11;

Measurement::Measurement(Digitizer *dig_, uint64_t averages, uint64_t batch, double part,
                         int second_oversampling, int K, const char *coil_address)
{
    dig = dig_;
    sampling_rate = static_cast<double>(dig->getSamplingRate());
    coil = new yokogawa_gs210(coil_address);
    segment_size = dig->getSegmentSize();
    batch_size = batch;
    setAveragesNumber(averages);
    notify_size = 2 * segment_size * batch_size;
    dig->handleError();
    dig->setTimeout(5000); // ms
    processor = new dsp(segment_size, batch_size, part, K, sampling_rate, second_oversampling);
    initializeBuffer();

    func = [this](int8_t *data) mutable
    { processor->compute(data); };

    int trace_length = processor->getTraceLength();

    test_input = new int8_t[notify_size * 2];
}

Measurement::Measurement(std::uintptr_t dig_handle, uint64_t averages, uint64_t batch, double part,
                         int second_oversampling, int K, const char *coil_address)
    : Measurement(new Digitizer(reinterpret_cast<void *>(dig_handle)), averages, batch, part,
                  second_oversampling, K, coil_address)
{
}

void Measurement::initializeBuffer()
{
    // Create the buffer in page-locked memory
    size_t buffersize = 4 * notify_size;
    processor->createBuffer(buffersize);
    dig->setBuffer(processor->getBuffer(), buffersize);
}

void Measurement::setCurrents(float wc, float oc)
{
    working_current = wc;
    offset_current = oc;
}

void Measurement::setAmplitude(int ampl)
{
    processor->setAmplitude(ampl);
}

/* Use frequency in GHz */
void Measurement::setIntermediateFrequency(float frequency)
{
    int oversampling = (int)std::round(1.25E+9f / dig->getSamplingRate());
    processor->setIntermediateFrequency(frequency, oversampling);
    hipDeviceSynchronize();
}

void Measurement::setAveragesNumber(uint64_t averages)
{
    segments_count = averages;
    iters_num = averages / batch_size;
    iters_done = 0;
}

void Measurement::setTapers(std::vector<stdvec> tapers)
{
    processor->setTapers(tapers);
}

void Measurement::setCalibration(float r, float phi, float offset_i, float offset_q)
{
    processor->setDownConversionCalibrationParameters(r, phi, offset_i, offset_q);
}

void Measurement::setFirwin(float left_cutoff, float right_cutoff)
{
    int oversampling = static_cast<int>(std::round(1.25E+9f / dig->getSamplingRate()));
    processor->setFirwin(left_cutoff, right_cutoff, oversampling);
    hipDeviceSynchronize();
}

void Measurement::measure()
{
    dig->prepareFifo(static_cast<unsigned long>(notify_size));
    dig->launchFifo(static_cast<unsigned long>(notify_size), iters_num, func, true);
    dig->stopFifo();
    iters_done += iters_num;
}

void Measurement::asyncCurrentSwitch()
{
    coil->set_current(working_current);
    setSubtractionTrace(getSubtractionData(), getSubtractionNoise());
    resetOutput();
    hipDeviceSynchronize();
}

void Measurement::measureWithCoil()
{
    coil->set_current(offset_current);
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
    dig->prepareFifo(notify_size);
    dig->launchFifo(notify_size, iters_num, func, true);
    iters_done += iters_num;

    uint64_t iters_delay = static_cast<size_t>(sampling_rate) / notify_size * 2;
    // auto a = std::async(std::launch::async, &Measurement::asyncCurrentSwitch, this);
    // dig->launchFifo(notify_size, iters_delay, func, false);
    // a.wait();

    std::thread t1(&Measurement::asyncCurrentSwitch, this);
    // std::thread t2 (&Digitizer::launchFifo, dig, notify_size, iters_delay, func, false);
    dig->launchFifo(notify_size, iters_delay, func, false);
    t1.join();
    // t2.join();
    // asyncCurrentSwitch();

    dig->launchFifo(notify_size, iters_num, func, true);
    iters_done += iters_num;
    dig->stopFifo();
}

void Measurement::measureTest()
{
    for (uint32_t i = 0; i < iters_num; i++)
        func(&test_input[0]);
    iters_done += iters_num;
}

void Measurement::setTestInput(py::buffer input)
{
    py::buffer_info info = input.request();
    if (info.ndim != 1)
        throw std::runtime_error("Number of dimensions must be one");
    if (static_cast<size_t>(info.size) < 2 * segment_size)
        throw std::runtime_error("Number of element in the imput array "
                                 "must be larger or equal to the two segment sizes");

    int8_t *input_ptr = (int8_t *)info.ptr;
    tiled_range<int8_t *> tiled_input(input_ptr, input_ptr + 2 * segment_size, batch_size);
    std::vector<int8_t> test_inp(test_input, test_input + 2 * notify_size);
    thrust::copy(tiled_input.begin(), tiled_input.end(), test_inp.begin());
}

stdvec_c Measurement::getMeanField()
{
    auto field_form_gpu = processor->getCumulativeField();
    return postprocess(field_form_gpu);
}

stdvec Measurement::getMeanPower()
{
    auto power_form_gpu = processor->getCumulativePower();
    return postprocess(power_form_gpu);
}

stdvec Measurement::postprocess(hostvec &data)
{
    using namespace thrust::placeholders;
    stdvec result(data.size());
    float divider = (iters_done > 0) ? static_cast<float>(iters_done) : 1.f;
    thrust::transform(data.begin(), data.end(), result.begin(), _1 / divider);
    return result;
}

stdvec_c Measurement::postprocess(hostvec_c &data)
{
    using namespace thrust::placeholders;
    stdvec_c result(data.size());
    float divider = (iters_done > 0) ? static_cast<float>(iters_done) : 1.f;
    thrust::transform(data.begin(), data.end(), result.begin(), _1 / divider);
    return result;
}

stdvec Measurement::getPSD()
{
    auto psd_spectrum = processor->getPowerSpectrum();
    return postprocess(psd_spectrum);
}

stdvec_c Measurement::getDataSpectrum()
{
    auto data_spectrum = processor->getDataSpectrum();
    return postprocess(data_spectrum);
}

stdvec_c Measurement::getNoiseSpectrum()
{
    auto noise_spectrum = processor->getNoiseSpectrum();
    return postprocess(noise_spectrum);
}

stdvec Measurement::getPeriodogram()
{
    auto periodogram = processor->getPeriodogram();
    return postprocess(periodogram);
}

std::vector<std::vector<std::complex<double>>> Measurement::getCorrelator()
{
    int len = processor->getOutSize();
    int side = processor->getTraceLength();

    hostvec_c result(len);
    std::vector<std::vector<std::complex<double>>> average_result(
        side, std::vector<std::complex<double>>(side));

    // Receive data from GPU
    processor->getCorrelator(result);

    // Divide the data by a number of traces measured
    int k = 0;
    tcf X((iters_done > 0) ? static_cast<float>(iters_done * batch_size) : 1.f, 0.f);
    for (int t1 = 0; t1 < side; t1++)
    {
        for (int t2 = t1; t2 < side; t2++)
        {
            k = t1 * side + t2;
            average_result[t1][t2] = std::complex<double>(result[k] / X);
            average_result[t2][t1] = std::conj(average_result[t1][t2]);
        }
    }
    return average_result;
}

stdvec_c Measurement::getRawCorrelator()
{
    int len = processor->getOutSize();
    int side = processor->getTraceLength();

    hostvec_c result(len);

    // Receive data from GPU
    processor->getCorrelator(result);

    return stdvec_c(result.begin(), result.end());
}

template <template <typename, typename...> class Container, typename T, typename... Args>
thrust::host_vector<T> Measurement::tile(const Container<T, Args...> &data, size_t N)
{
    // data : vector to tile
    // N : how much to tile
    using iter_t = typename Container<T, Args...>::const_iterator;
    thrust::host_vector<T> tiled_data(data.size() * N);
    tiled_range<iter_t> tiled_iter(data.begin(), data.end(), N);
    thrust::copy(tiled_iter.begin(), tiled_iter.end(), tiled_data.begin());
    return tiled_data;
}

void Measurement::setSubtractionTrace(stdvec_c trace, stdvec_c offsets)
{
    hostvec_c average = tile(trace, batch_size);
    hostvec_c average_offsets = tile(offsets, batch_size);
    processor->setSubtractionTrace(average, average_offsets);
}

stdvec_c Measurement::getSubtractionData()
{
    auto subtr_data_from_gpu = processor->getCumulativeSubtrData();
    return postprocess(subtr_data_from_gpu);
}

stdvec_c Measurement::getSubtractionNoise()
{
    auto subtr_noise_from_gpu = processor->getCumulativeSubtrNoise();
    return postprocess(subtr_noise_from_gpu);
}

py::tuple Measurement::getSubtractionTrace()
{
    auto len = processor->getTotalLength();
    hostvec_c subtraction_trace(len);
    hostvec_c subtraction_offs(len);
    processor->getSubtractionTrace(subtraction_trace, subtraction_offs);
    return py::make_tuple(stdvec_c(subtraction_trace.begin(), subtraction_trace.end()),
                          stdvec_c(subtraction_offs.begin(), subtraction_offs.end()));
}

std::vector<std::vector<float>> Measurement::getDPSSTapers()
{
    auto tapers = processor->getDPSSTapers();

    size_t num_rows = tapers.size();
    size_t num_cols = (num_rows > 0) ? tapers[0].size() : 0;

    std::vector<std::vector<float>> result(num_rows);
    for (size_t i = 0; i < num_rows; ++i)
    {
        result[i].resize(num_cols);
        std::copy(tapers[i].begin(), tapers[i].end(), result[i].begin());
    }

    return result;
}

void Measurement::reset()
{
    this->resetOutput();
    processor->resetSubtractionTrace();
}

void Measurement::resetOutput()
{
    iters_done = 0;
    processor->resetOutput();
}

void Measurement::free()
{
    delete processor;
    delete dig;
    processor = NULL;
    dig = NULL;
    delete[] test_input;
}

Measurement::~Measurement()
{
    if ((processor != NULL) || (dig != NULL))
        this->free();
}