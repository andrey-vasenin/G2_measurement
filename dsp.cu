#include "hip/hip_runtime.h"
//
// Created by andrei on 3/27/21.
//

#include "dsp.cuh"
#include "dsp_functors.cuh"
#include <iostream>
#include <string>
#include <stdexcept>
#include <vector>
#include <npp.h>
#include <nppcore.h>
#include <nppdefs.h>
#include <npps.h>
#include <complex>
#include <hipblas.h>
#include <cmath>
#include <numeric>
#include "strided_range.cuh"
#include "tiled_range.cuh"
#include <thrust/complex.h>
#include <thrust/transform.h>
#include <thrust/tabulate.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/zip_function.h>
#include <thrust/iterator/constant_iterator.h>


inline void check_cufft_error(hipfftResult cufft_err, std::string &&msg)
{
#ifdef NDEBUG

    if (cufft_err != HIPFFT_SUCCESS)
        throw std::runtime_error(msg);

#endif // NDEBUG
}

inline void check_cublas_error(hipblasStatus_t err, std::string &&msg)
{
#ifdef NDEBUG

    if (err != HIPBLAS_STATUS_SUCCESS)
        throw std::runtime_error(msg);

#endif // NDEBUG
}

inline void check_npp_error(NppStatus err, std::string &&msg)
{
#ifdef NDEBUG
    if (err != NPP_SUCCESS)
        throw std::runtime_error(msg);
#endif // NDEBUG
}

template <typename T> 
inline void print_vector(thrust::device_vector<T> & vec, int n) 
{
    hipDeviceSynchronize();
    thrust::copy(vec.begin(), vec.begin() + n, std::ostream_iterator<T>(std::cout, " "));
    std::cout << std::endl;
}

inline void print_gpu_buff(gpubuf vec, int n)
{
    hipDeviceSynchronize();
    thrust::copy(vec.begin(), vec.begin() + n, std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;
}


// DSP constructor
dsp::dsp(size_t len, uint64_t n, double part, int K_,
    double samplerate, int second_oversampling) : 
    trace_length{static_cast<size_t>(std::round((double)len * part))}, // Length of a signal or noise trace
    batch_size{n},                                    // Number of segments in a buffer (same: number of traces in data)
    total_length{batch_size * trace_length},
    oversampling{ second_oversampling },
    resampled_trace_length{ trace_length / oversampling },
    resampled_total_length{ total_length / oversampling },
    out_size{trace_length * trace_length},
    trace1_start{0},       // Start of the signal data
    trace2_start{len / 2}, // Start of the noise data
    pitch{len}           // Segment length in a buffer
                                       
{
    //firwin.resize(total_length); // GPU memory for the filtering window
    subtraction_trace.resize(total_length);
    subtraction_offs.resize(total_length);
    thrust::fill(subtraction_trace.begin(), subtraction_trace.end(), tcf(0.f));
    downconversion_coeffs.resize(total_length);

    // Setup multitaper
    K = K_;
    tapers.resize(K);

    // Streams
    for (int i = 0; i < num_streams; i++)
    {
        // Create streams for parallel data processing
        handleError(hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));
        check_npp_error(nppGetStreamContext(&streamContexts[i]), "Npp Error GetStreamContext");
        streamContexts[i].hStream = streams[i];

        // Allocate arrays on GPU for every stream
        gpu_data_buf[i].resize(2 * total_length);
        gpu_noise_buf[i].resize(2 * total_length);
        data[i].resize(total_length);
        data_resampled[i].resize(resampled_total_length);
        subtraction_data[i].resize(total_length);
        noise[i].resize(total_length);
        noise_resampled[i].resize(resampled_total_length);
        subtraction_noise[i].resize(total_length);
        power[i].resize(total_length);
        field[i].resize(total_length);
        //out[i].resize(out_size);
        taperedData[i].resize(resampled_total_length);
        taperedNoise[i].resize(resampled_total_length);
        data_fft[i].resize(resampled_total_length);
        noise_fft[i].resize(resampled_total_length);
        spectrum[i].resize(resampled_total_length);
        periodogram[i].resize(total_length);

        // Initialize cuFFT plans
        check_cufft_error(hipfftPlan1d(&plans[i], trace_length, HIPFFT_C2C, batch_size),
                          "Error initializing cuFFT plan\n");
        check_cufft_error(hipfftPlan1d(&multitaper_plans[i], resampled_trace_length, HIPFFT_C2C, batch_size),
            "Error initializing cuFFT plan\n");

        // Assign streams to cuFFT plans
        check_cufft_error(hipfftSetStream(plans[i], streams[i]),
                          "Error assigning a stream to a cuFFT plan\n");
        check_cufft_error(hipfftSetStream(multitaper_plans[i], streams[i]),
            "Error assigning a stream to a cuFFT plan\n");

        // Initialize cuBLAS
        check_cublas_error(hipblasCreate(&cublas_handles[i]),
                           "Error initializing a cuBLAS handle\n");
        check_cublas_error(hipblasCreate(&cublas_handles2[i]),
                           "Error initializing a cuBLAS handle\n");

        // Assign streams to cuBLAS handles
        check_cublas_error(hipblasSetStream(cublas_handles[i], streams[i]),
                           "Error assigning a stream to a cuBLAS handle\n");
        check_cublas_error(hipblasSetStream(cublas_handles2[i], streams[i]),
                           "Error assigning a stream to a cuBLAS handle\n");
    }
    resetOutput();
    resetSubtractionTrace();
}

// DSP destructor
dsp::~dsp()
{
    deleteBuffer();
    for (int i = 0; i < num_streams; i++)
    {
        // Destroy cuBLAS
        hipblasDestroy(cublas_handles[i]);
        hipblasDestroy(cublas_handles2[i]);

        // Destroy cuFFT plans
        hipfftDestroy(plans[i]);
        hipfftDestroy(multitaper_plans[i]);

        // Destroy GPU streams
        handleError(hipStreamDestroy(streams[i]));
    }
}

// Creates a rectangular window with specified cutoff frequencies for the further usage in a filter
void dsp::setFirwin(float cutoff_l, float cutoff_r, int oversampling)
{
    using namespace std::complex_literals;
    hostvec_c hFirwin(total_length);
    float fs = 1250.f / (float)oversampling;
    int l_idx = (int)std::roundf((float)trace_length / fs * cutoff_l);
    int r_idx = (int)std::roundf((float)trace_length / fs * cutoff_r);
    for (int i = 0; i < total_length; i++)
    {
        int j = i % trace_length;
        hFirwin[i] = ((j < l_idx) || (j > r_idx)) ? 0if : 1.0f + 0if;
    }
    firwin = hFirwin;
}

// Error handler
void dsp::handleError(hipError_t err)
{
    if (err != hipSuccess)
    {
        std::string name = hipGetErrorName(err);
        std::string text = hipGetErrorString(err);
        throw std::runtime_error(name + ": " + text);
    }
}

void dsp::createBuffer(size_t size)
{
    this->handleError(hipHostMalloc((void**)&buffer, size));
}

void dsp::deleteBuffer() {
    this->handleError(hipHostFree(buffer));
};

void dsp::setIntermediateFrequency(float frequency, int oversampling)
{
    const float pi = std::acos(-1.f);
    float ovs = static_cast<float>(oversampling);
    hostvec_c hDownConv(total_length);
    thrust::tabulate(hDownConv.begin(), hDownConv.end(),
        [=] __host__ (int i) -> tcf {
            float t = 0.8 * ovs * static_cast<float>(i % trace_length);
            return thrust::exp(tcf(0, -2 * pi * frequency * t));
        });
    downconversion_coeffs = hDownConv;
}

void dsp::downconvert(gpuvec_c &data, int stream_num)
{
    // thrust::transform(thrust::cuda::par_nosync.on(stream), data.begin(), data.end(), downconversion_coeffs.begin(), data.begin(), downconv_functor());
    Npp32fc* src = reinterpret_cast<Npp32fc*>(thrust::raw_pointer_cast(data.data()));
    const Npp32fc* coef = reinterpret_cast<const Npp32fc*>(thrust::raw_pointer_cast(downconversion_coeffs.data()));
    auto status = nppsMul_32fc_I_Ctx(coef, src, data.size(), streamContexts[stream_num]);
    check_npp_error(status, "Error with downconversion");
}

void dsp::setDownConversionCalibrationParameters(float r, float phi,
    float offset_i, float offset_q)
{
    a_qi = std::tan(phi);
    a_qq = 1 / (r * std::cos(phi));
    c_i = offset_i;
    c_q = offset_q;
}

// Applies down-conversion calibration to traces
void dsp::applyDownConversionCalibration(gpuvec_c& data, hipStream_t &stream)
{
    auto sync_exec_policy = thrust::cuda::par_nosync.on(stream);
    thrust::for_each(sync_exec_policy, data.begin(), data.end(), calibration_functor(a_qi, a_qq, c_i, c_q));
}

hostbuf dsp::getBuffer()
{
    return buffer;
}

// Fills with zeros the arrays for cumulative field and power in the GPU memory
void dsp::resetOutput()
{
    for (int i = 0; i < num_streams; i++)
    {
        //thrust::fill(out[i].begin(), out[i].end(), tcf(0));
        thrust::fill(field[i].begin(), field[i].end(), tcf(0));
        thrust::fill(power[i].begin(), power[i].end(), 0.f);
        thrust::fill(spectrum[i].begin(), spectrum[i].end(), 0.f);
        thrust::fill(periodogram[i].begin(), periodogram[i].end(), 0.f);
        thrust::fill(data_fft[i].begin(), data_fft[i].end(), tcf(0));
        thrust::fill(noise_fft[i].begin(), noise_fft[i].end(), tcf(0));
        thrust::fill(subtraction_data[i].begin(), subtraction_data[i].end(), tcf(0));
        thrust::fill(subtraction_noise[i].begin(), subtraction_noise[i].end(), tcf(0));
    }
}

void dsp::compute(const hostbuf buffer_ptr)
{
    const int stream_num = semaphore;
    switchStream();
    loadDataToGPUwithPitchAndOffset(buffer_ptr, gpu_data_buf[stream_num], pitch, trace1_start, stream_num);
    loadDataToGPUwithPitchAndOffset(buffer_ptr, gpu_noise_buf[stream_num], pitch, trace2_start, stream_num);
    convertDataToMillivolts(data[stream_num], gpu_data_buf[stream_num], streams[stream_num]); // error is here
    convertDataToMillivolts(noise[stream_num], gpu_noise_buf[stream_num], streams[stream_num]);
    applyDownConversionCalibration(data[stream_num], streams[stream_num]);
    applyDownConversionCalibration(noise[stream_num], streams[stream_num]);
    applyFilter(data[stream_num], firwin, stream_num);
    applyFilter(noise[stream_num], firwin, stream_num);
    downconvert(data[stream_num], stream_num);
    downconvert(noise[stream_num], stream_num);

    subtractDataFromOutput(subtraction_trace, data[stream_num], stream_num);
    subtractDataFromOutput(subtraction_offs, noise[stream_num], stream_num);

    addDataToOutput(data[stream_num], subtraction_data[stream_num], stream_num);
    addDataToOutput(noise[stream_num], subtraction_noise[stream_num], stream_num);

    calculateField(data[stream_num], noise[stream_num],
        field[stream_num], streams[stream_num]);
    calculatePower(data[stream_num], noise[stream_num], power[stream_num], streams[stream_num]);
    //calculateG1(data_calibrated[stream_num], noise_calibrated[stream_num],
    //    out[stream_num], cublas_handles[stream_num]);
    resample(data[stream_num], data_resampled[stream_num], streams[stream_num]);
    resample(noise[stream_num], noise_resampled[stream_num], streams[stream_num]);
    calculateMultitaperSpectrum(data_resampled[stream_num], noise_resampled[stream_num],
        data_fft[stream_num], noise_fft[stream_num], spectrum[stream_num], stream_num);
    calculatePeriodogram(data[stream_num], noise[stream_num],
        periodogram[stream_num], stream_num);
}

// This function uploads data from the specified section of a buffer array to the GPU memory
void dsp::loadDataToGPUwithPitchAndOffset(const hostbuf buffer_ptr,
    gpubuf& gpu_buf, size_t pitch, size_t offset, int stream_num)
{
    size_t width = 2 * size_t(trace_length) * sizeof(int8_t);
    size_t src_pitch = 2 * pitch * sizeof(int8_t);
    size_t dst_pitch = width;
    size_t shift = 2 * offset;
    handleError(hipMemcpy2DAsync(thrust::raw_pointer_cast(gpu_buf.data()), dst_pitch,
                                  static_cast<const void*>(buffer_ptr + shift), src_pitch, width, batch_size,
                                  hipMemcpyHostToDevice, streams[stream_num]));
}

// Converts bytes into 32-bit floats with mV dimensionality
void dsp::convertDataToMillivolts(gpuvec_c& data, const gpubuf& gpu_buf, const hipStream_t &stream)
{
    using iter = gpubuf::const_iterator;
    strided_range<iter> channelI(gpu_buf.begin(), gpu_buf.end(), 2);
    strided_range<iter> channelQ(gpu_buf.begin() + 1, gpu_buf.end(), 2);
    thrust::transform(thrust::cuda::par_nosync.on(stream),
        channelI.begin(), channelI.end(), channelQ.begin(), data.begin(), millivolts_functor(scale));
}

// Applies the filter with the specified window to the data using FFT convolution
void dsp::applyFilter(gpuvec_c &data, const gpuvec_c &window, int stream_num)
{
    // Step 1. Take FFT of each segment
    hipfftComplex *cufft_data = reinterpret_cast<hipfftComplex *>(thrust::raw_pointer_cast(data.data()));
    auto cufftstat = hipfftExecC2C(plans[stream_num], cufft_data, cufft_data, HIPFFT_FORWARD);
    check_cufft_error(cufftstat, "Error executing cufft");
    // Step 2. Multiply each segment by a window
    thrust::transform(thrust::cuda::par_nosync.on(streams[stream_num]),
        data.begin(), data.end(), window.begin(), data.begin(), thrust::multiplies<tcf>());
    // Step 3. Take inverse FFT of each segment
    hipfftExecC2C(plans[stream_num], cufft_data, cufft_data, HIPFFT_BACKWARD);
    check_cufft_error(cufftstat, "Error executing cufft");
    // Step 4. Normalize the FFT for the output to equal the input
    thrust::transform(thrust::cuda::par_nosync.on(streams[stream_num]),
        data.begin(), data.end(), thrust::constant_iterator<tcf>(1.f / static_cast<float>(trace_length)),
        data.begin(), thrust::multiplies<tcf>());
}

// Sums newly processed data with previous data for averaging
void dsp::addDataToOutput(const gpuvec_c &data, gpuvec_c &output, int stream_num)
{
    const Npp32fc* src = reinterpret_cast<const Npp32fc*>(thrust::raw_pointer_cast(data.data()));
    Npp32fc* dst = reinterpret_cast<Npp32fc*>(thrust::raw_pointer_cast(output.data()));
    auto status = nppsAdd_32fc_I_Ctx(src, dst, data.size(), streamContexts[stream_num]);
    check_npp_error(status, "Error adding two vectors");
}

// Subtracts newly processed data from previous data
void dsp::subtractDataFromOutput(const gpuvec_c& data, gpuvec_c& output, int stream_num)
{
    const Npp32fc* src = reinterpret_cast<const Npp32fc*>(thrust::raw_pointer_cast(data.data()));
    Npp32fc* dst = reinterpret_cast<Npp32fc*>(thrust::raw_pointer_cast(output.data()));
    auto status = nppsSub_32fc_I_Ctx(src, dst, data.size(), streamContexts[stream_num]);
    check_npp_error(status, "Error subtracting two vectors");
    /*thrust::transform(thrust::cuda::par_nosync.on(stream), output.begin(), output.end(), data.begin(),
        output.begin(), thrust::minus<tcf>());*/
}

// Calculates the field from the data in the GPU memory
void dsp::calculateField(const gpuvec_c& data, const gpuvec_c& noise, gpuvec_c& output, const hipStream_t &stream)
{
    thrust::for_each(thrust::cuda::par_nosync.on(stream),
        thrust::make_zip_iterator(data.begin(), noise.begin(), output.begin()),
        thrust::make_zip_iterator(data.end(), noise.end(), output.end()),
        thrust::make_zip_function(field_functor()));
}

void dsp::resample(const gpuvec_c& traces, gpuvec_c& resampled_traces, const hipStream_t& stream)
{
    using iter = gpuvec_c::const_iterator;
    switch (oversampling)
    {
    case 1:
        thrust::copy(thrust::cuda::par_nosync.on(stream), traces.begin(), traces.end(), resampled_traces.begin());
        break;
    case 2:
    {
        strided_range<iter> t1(traces.begin(), traces.end(), oversampling);
        strided_range<iter> t2(traces.begin() + 1, traces.end(), oversampling);
        auto beginning = thrust::make_zip_iterator(t1.begin(), t2.begin());
        auto end = thrust::make_zip_iterator(t1.end(), t2.end());
        thrust::transform(thrust::cuda::par_nosync.on(stream), beginning, end, resampled_traces.begin(),
            thrust::make_zip_function(downsample2_functor()));
        break;
    }
    case 4:
    {
        strided_range<iter> t1(traces.begin(), traces.end(), oversampling);
        strided_range<iter> t2(traces.begin() + 1, traces.end(), oversampling);
        strided_range<iter> t3(traces.begin() + 2, traces.end(), oversampling);
        strided_range<iter> t4(traces.begin() + 3, traces.end(), oversampling);
        auto beginning = thrust::make_zip_iterator(t1.begin(), t2.begin(), t3.begin(), t4.begin());
        auto end = thrust::make_zip_iterator(t1.end(), t2.end(), t3.end(), t4.end());
        thrust::transform(thrust::cuda::par_nosync.on(stream), beginning, end, resampled_traces.begin(),
            thrust::make_zip_function(downsample4_functor()));
        break;
    }
    default:
        throw std::runtime_error("Unsupported second oversampling");
    }
}

// Calculates the power from the data in the GPU memory
void dsp::calculatePower(const gpuvec_c& data, const gpuvec_c& noise, gpuvec& output, const hipStream_t& stream)
{
    thrust::for_each(thrust::cuda::par_nosync.on(stream),
        thrust::make_zip_iterator(data.begin(), noise.begin(), output.begin()),
        thrust::make_zip_iterator(data.end(), noise.end(), output.end()),
        thrust::make_zip_function(power_functor()));
}

void dsp::calculateG1(gpuvec_c& data, gpuvec_c& noise, gpuvec_c& output, hipblasHandle_t &handle)
{
    using namespace std::string_literals;

    const float alpha_data = 1;   // this alpha multiplies the result to be added to the output
    const float alpha_noise = -1; // this alpha multiplies the result to be added to the output
    const float beta = 1;
    // Compute correlation for the signal and add it to the output
    auto cublas_status = hipblasCherk(handle,
                                     HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, trace_length, batch_size,
                                     &alpha_data, reinterpret_cast<hipComplex *>(thrust::raw_pointer_cast(data.data())), trace_length,
                                     &beta, reinterpret_cast<hipComplex *>(thrust::raw_pointer_cast(output.data())), trace_length);
    // Check for errors
    check_cublas_error(cublas_status,
        "Error of rank-1 update (data) with code #"s + std::to_string(cublas_status));
    // Compute correlation for the noise and subtract it from the output
    cublas_status = hipblasCherk(handle,
                                HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, trace_length, batch_size,
                                &alpha_noise, reinterpret_cast<hipComplex *>(thrust::raw_pointer_cast(noise.data())), trace_length,
                                &beta, reinterpret_cast<hipComplex *>(thrust::raw_pointer_cast(output.data())), trace_length);
    // Check for errors
    check_cublas_error(cublas_status,
        "Error of rank-1 update (noise) with code #"s + std::to_string(cublas_status));
}

void dsp::calculatePeriodogram(gpuvec_c& data, gpuvec_c& noise, gpuvec& output, int stream_num)
{
    hipfftComplex* cufft_data = reinterpret_cast<hipfftComplex*>(thrust::raw_pointer_cast(data.data()));
    auto cufftstat1 = hipfftExecC2C(plans[stream_num], cufft_data, cufft_data, HIPFFT_FORWARD);
    check_cufft_error(cufftstat1, "Error executing cufft");

    hipfftComplex* cufft_noise = reinterpret_cast<hipfftComplex*>(thrust::raw_pointer_cast(noise.data()));
    auto cufftstat2 = hipfftExecC2C(plans[stream_num], cufft_noise, cufft_noise, HIPFFT_FORWARD);
    check_cufft_error(cufftstat2, "Error executing cufft");

    thrust::for_each(thrust::cuda::par_nosync.on(streams[stream_num]),
        thrust::make_zip_iterator(data.begin(), noise.begin(), output.begin()),
        thrust::make_zip_iterator(data.end(), noise.end(), output.end()),
        thrust::make_zip_function(power_functor()));
}

void dsp::calculateMultitaperSpectrum(const gpuvec_c& data, const gpuvec_c& noise, gpuvec_c& signal_field_spectra,
    gpuvec_c& noise_field_spectra, gpuvec& power_spectra, int stream_num)
{
    for (size_t i = 0; i < K; ++i) {
        // 1. Windowing the Signal with Tapers
        const Npp32f* src1_t = reinterpret_cast<const Npp32f*>(thrust::raw_pointer_cast(tapers[i].data()));
        const Npp32fc* src2_d = reinterpret_cast<const Npp32fc*>(thrust::raw_pointer_cast(data.data()));
        Npp32fc* dst_d = reinterpret_cast<Npp32fc*>(thrust::raw_pointer_cast(taperedData[stream_num].data()));
        nppsMul_32f32fc_Ctx(src1_t, src2_d, dst_d, data.size(), streamContexts[stream_num]);
        const Npp32fc* src2_n = reinterpret_cast<const Npp32fc*>(thrust::raw_pointer_cast(noise.data()));
        Npp32fc* dst_n = reinterpret_cast<Npp32fc*>(thrust::raw_pointer_cast(taperedNoise[stream_num].data()));
        nppsMul_32f32fc_Ctx(src1_t, src2_n, dst_n, data.size(), streamContexts[stream_num]);
        // 2. FFT
        auto cufft_tapered_data = reinterpret_cast<hipfftComplex*>(thrust::raw_pointer_cast(taperedData[stream_num].data()));
        auto cufft_tapered_noise = reinterpret_cast<hipfftComplex*>(thrust::raw_pointer_cast(taperedNoise[stream_num].data()));
        hipfftExecC2C(multitaper_plans[stream_num], cufft_tapered_data, cufft_tapered_data, HIPFFT_FORWARD);
        hipfftExecC2C(multitaper_plans[stream_num], cufft_tapered_noise, cufft_tapered_noise, HIPFFT_FORWARD);
        // 3. Compute Field Spectra
        addDataToOutput(taperedData[stream_num], signal_field_spectra, stream_num);
        addDataToOutput(taperedNoise[stream_num], noise_field_spectra, stream_num);
        // 4. Compute Power Spectra
        thrust::for_each(thrust::cuda::par_nosync.on(streams[stream_num]),
            thrust::make_zip_iterator(taperedData[stream_num].begin(), taperedNoise[stream_num].begin(), power_spectra.begin()),
            thrust::make_zip_iterator(taperedData[stream_num].end(), taperedNoise[stream_num].end(), power_spectra.end()),
            thrust::make_zip_function(power_functor()));
    }

}

template<typename T>
thrust::host_vector<T> dsp::getCumulativeTrace(const thrust::device_vector<T>* traces)
{
    thrust::device_vector<T> tmp(traces->size(), T(0));
    this->handleError(hipDeviceSynchronize());
    for (int i = 0; i < num_streams; i++)
        thrust::transform(traces[i].begin(), traces[i].end(), tmp.begin(), tmp.begin(), thrust::plus<T>());
    size_t N = traces->size() / batch_size;
    thrust::host_vector<T> host_trace(N);
    using iter = typename thrust::device_vector<T>::iterator;
    for (size_t j = 0; j < N; ++j) {
        strided_range<iter> tmp_iter(tmp.begin() + j, tmp.end(), N);
        T el = thrust::reduce(tmp_iter.begin(), tmp_iter.end(), T(0), thrust::plus<T>());
        host_trace[j] = el / T(batch_size);
    }
    return host_trace;
}

// Returns the average value
void dsp::getCorrelator(hostvec_c& result)
{
    gpuvec_c c(out[0].size(), tcf(0));
    this->handleError(hipDeviceSynchronize());
    for (int i = 0; i < num_streams; i++)
        thrust::transform(out[i].begin(), out[i].end(), c.begin(), c.begin(), thrust::plus<tcf>());
    result = c;
}

// Returns the cumulative power
hostvec dsp::getCumulativePower()
{
    return getCumulativeTrace(power);
}

hostvec dsp::getPowerSpectrum()
{
    return getCumulativeTrace(spectrum);
}

hostvec dsp::getPeriodogram()
{
    return getCumulativeTrace(periodogram);
}

hostvec_c dsp::getDataSpectrum()
{
    return getCumulativeTrace(data_fft);
}

hostvec_c dsp::getNoiseSpectrum()
{
    return getCumulativeTrace(noise_fft);
}

// Returns the cumulative field
hostvec_c dsp::getCumulativeField()
{
    return getCumulativeTrace(field);
}

hostvec_c dsp::getCumulativeSubtrData()
{
    return getCumulativeTrace(subtraction_data);
}

hostvec_c dsp::getCumulativeSubtrNoise()
{
    return getCumulativeTrace(subtraction_noise);
}

// Returns the useful length of the data in a segment
// (trace is assumed complex valued)
int dsp::getTraceLength()
{
    return trace_length;
}

// Returns the total length of the data comprised of several segments
// (trace is assumed complex valued)
int dsp::getTotalLength()
{
    return total_length;
}

int dsp::getOutSize()
{
    return out_size;
}

void dsp::setAmplitude(int ampl)
{
    scale = static_cast<float>(ampl) / 128.f;
}

void dsp::setSubtractionTrace(hostvec_c &trace, hostvec_c& offsets)
{
    subtraction_trace = trace;
    subtraction_offs = offsets;
}

void dsp::getSubtractionTrace(hostvec_c &trace, hostvec_c& offsets)
{
    trace = subtraction_trace;
    offsets = subtraction_offs;
}

void dsp::resetSubtractionTrace()
{
    thrust::fill(subtraction_trace.begin(), subtraction_trace.end(), tcf(0));
    thrust::fill(subtraction_offs.begin(), subtraction_offs.end(), tcf(0));
}

void dsp::setTapers(std::vector<stdvec> h_tapers)
{
    if (h_tapers.size() != K)
        throw std::runtime_error("Tapers number is not equal K");
    if (h_tapers[0].size() != resampled_trace_length)
        throw std::runtime_error("Taper length is not equal resampled_trace_length");
    for (size_t i = 0; i < K; i++)
    {
        hostvec h_taper_batched(resampled_total_length);
        tiled_range<stdvec::iterator> tiled_taper_range(h_tapers[i].begin(), h_tapers[i].end(), batch_size);
        thrust::copy(tiled_taper_range.begin(), tiled_taper_range.end(), h_taper_batched.begin());
        tapers[i] = h_taper_batched;
    }
}

std::vector<hostvec> dsp::getDPSSTapers()
{
    std::vector<hostvec> h_tapers(K);
    for (int i = 0; i < K; i++)
    {
        h_tapers[i] = tapers[i];
    }
    return h_tapers;
}